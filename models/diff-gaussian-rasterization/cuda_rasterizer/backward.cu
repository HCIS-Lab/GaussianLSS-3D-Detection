#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "backward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

// Backward version of INVERSE 2D covariance matrix computation
// (due to length launched as separate kernel before other 
// backward steps contained in preprocess)

__global__ void computeCov2DBEVCUDA(int P,
	const float3* means,
	const int* radii,
	const float* cov3Ds,
	const float* view_matrix,
	const float* dL_dconics,
	// float3* dL_dmeans,
	float* dL_dcov)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P || !(radii[idx] > 0))
		return;

	// Reading location of 3D covariance for this Gaussian
	const float* cov3D = cov3Ds + 6 * idx;

	// Fetch gradients, recompute 2D covariance and relevant 
	// intermediate forward results needed in the backward.
	// float3 mean = means[idx];
	float3 dL_dconic = { dL_dconics[4 * idx], dL_dconics[4 * idx + 1], dL_dconics[4 * idx + 3] };

	// In BEV plane: 
	// - X axis = -World Y axis
	// - Y axis = -World X axis
	
	// Extract scaling factors for the BEV plane
	const float x_scale = view_matrix[0];  // corresponds to world X
	const float y_scale = view_matrix[5];  // corresponds to world Y

	// Compute 2D BEV covariance with view transformation
	// Since we swapped axes, we adjust accordingly.
	// Covariance elements in the BEV plane, swapping axes and applying scaling.
	float3 cov2D = { float(cov3D[0] * x_scale * x_scale + 0.3f),     // cov3D[3] -> world y, maps to BEV x
					 float(cov3D[1] * x_scale * y_scale),     // cov3D[1] -> cov3D xy term
					 float(cov3D[3] * y_scale * y_scale + 0.3f) };   // cov3D[0] -> world x, maps to BEV y
	
	// Use helper variables for 2D covariance entries. More compact.
	float a = cov2D.x;
	float b = cov2D.y;
	float c = cov2D.z;

	// Determinant of 2D covariance matrix
	float denom = a * c - b * b;
	float denom2inv = 1.0f / ((denom * denom) + 0.0000001f);

	// Gradients of loss w.r.t. 2D covariance entries
	float dL_da = 0, dL_db = 0, dL_dc = 0;
	if (denom2inv != 0)
	{
		dL_da = denom2inv * (-c * c * dL_dconic.x + 2 * b * c * dL_dconic.y + (denom - a * c) * dL_dconic.z);
		dL_dc = denom2inv * (-a * a * dL_dconic.z + 2 * a * b * dL_dconic.y + (denom - a * c) * dL_dconic.x);
		dL_db = denom2inv * 2 * (b * c * dL_dconic.x - (denom + 2 * b * b) * dL_dconic.y + a * b * dL_dconic.z);

		// Update gradients w.r.t. 3D covariance matrix (Vrk) entry, using the new axis mapping.
		dL_dcov[6 * idx + 0] = (x_scale * x_scale * dL_da);     // Cov3D[3], maps to cov2D[0] (world y to BEV x)
		dL_dcov[6 * idx + 1] = x_scale * y_scale * dL_db;    // Cov3D[1], maps to cov2D[1] (off-diagonal)
		dL_dcov[6 * idx + 3] = (y_scale * y_scale * dL_dc);      // Cov3D[0], maps to cov2D[2] (world x to BEV y)
		dL_dcov[6 * idx + 2] = 0; // X Z
		dL_dcov[6 * idx + 4] = 0; // Y Z
		dL_dcov[6 * idx + 5] = 0; // Z Z
	}
	else
	{
		// If the determinant is too small or zero, set gradients to zero
		for (int i = 0; i < 6; i++)
			dL_dcov[6 * idx + i] = 0;
	}
}


// Backward pass for the conversion of scale and rotation to a 
// 3D covariance matrix for each Gaussian. 
__device__ void computeCov3D(int idx, const glm::vec3 scale, float mod, const glm::vec4 rot, const float* dL_dcov3Ds, glm::vec3* dL_dscales, glm::vec4* dL_drots)
{
	// Recompute (intermediate) results for the 3D covariance computation.
	glm::vec4 q = rot;// / glm::length(rot);
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	glm::mat3 S = glm::mat3(1.0f);

	glm::vec3 s = mod * scale;
	S[0][0] = s.x;
	S[1][1] = s.y;
	S[2][2] = s.z;

	glm::mat3 M = S * R;

	const float* dL_dcov3D = dL_dcov3Ds + 6 * idx;

	glm::vec3 dunc(dL_dcov3D[0], dL_dcov3D[3], dL_dcov3D[5]);
	glm::vec3 ounc = 0.5f * glm::vec3(dL_dcov3D[1], dL_dcov3D[2], dL_dcov3D[4]);

	// Convert per-element covariance loss gradients to matrix form
	glm::mat3 dL_dSigma = glm::mat3(
		dL_dcov3D[0], 0.5f * dL_dcov3D[1], 0.5f * dL_dcov3D[2],
		0.5f * dL_dcov3D[1], dL_dcov3D[3], 0.5f * dL_dcov3D[4],
		0.5f * dL_dcov3D[2], 0.5f * dL_dcov3D[4], dL_dcov3D[5]
	);

	// Compute loss gradient w.r.t. matrix M
	// dSigma_dM = 2 * M
	glm::mat3 dL_dM = 2.0f * M * dL_dSigma;

	glm::mat3 Rt = glm::transpose(R);
	glm::mat3 dL_dMt = glm::transpose(dL_dM);

	// Gradients of loss w.r.t. scale
	glm::vec3* dL_dscale = dL_dscales + idx;
	dL_dscale->x = glm::dot(Rt[0], dL_dMt[0]);
	dL_dscale->y = glm::dot(Rt[1], dL_dMt[1]);
	dL_dscale->z = glm::dot(Rt[2], dL_dMt[2]);

	dL_dMt[0] *= s.x;
	dL_dMt[1] *= s.y;
	dL_dMt[2] *= s.z;

	// Gradients of loss w.r.t. normalized quaternion
	glm::vec4 dL_dq;
	dL_dq.x = 2 * z * (dL_dMt[0][1] - dL_dMt[1][0]) + 2 * y * (dL_dMt[2][0] - dL_dMt[0][2]) + 2 * x * (dL_dMt[1][2] - dL_dMt[2][1]);
	dL_dq.y = 2 * y * (dL_dMt[1][0] + dL_dMt[0][1]) + 2 * z * (dL_dMt[2][0] + dL_dMt[0][2]) + 2 * r * (dL_dMt[1][2] - dL_dMt[2][1]) - 4 * x * (dL_dMt[2][2] + dL_dMt[1][1]);
	dL_dq.z = 2 * x * (dL_dMt[1][0] + dL_dMt[0][1]) + 2 * r * (dL_dMt[2][0] - dL_dMt[0][2]) + 2 * z * (dL_dMt[1][2] + dL_dMt[2][1]) - 4 * y * (dL_dMt[2][2] + dL_dMt[0][0]);
	dL_dq.w = 2 * r * (dL_dMt[0][1] - dL_dMt[1][0]) + 2 * x * (dL_dMt[2][0] + dL_dMt[0][2]) + 2 * y * (dL_dMt[1][2] + dL_dMt[2][1]) - 4 * z * (dL_dMt[1][1] + dL_dMt[0][0]);

	// Gradients of loss w.r.t. unnormalized quaternion
	float4* dL_drot = (float4*)(dL_drots + idx);
	*dL_drot = float4{ dL_dq.x, dL_dq.y, dL_dq.z, dL_dq.w };//dnormvdv(float4{ rot.x, rot.y, rot.z, rot.w }, float4{ dL_dq.x, dL_dq.y, dL_dq.z, dL_dq.w });
}

// Backward pass of the preprocessing steps, except
// for the covariance computation and inversion
// (those are handled by a previous kernel call)
template<int C>
__global__ void preprocessCUDA(
	int P, int D, int M,
	const float3* means,
	const int* radii,
	const float* shs,
	const bool* clamped,
	const glm::vec3* scales,
	const glm::vec4* rotations,
	const float scale_modifier,
	const float* proj,
	// const glm::vec3* campos,
	const float3* dL_dmean2D,
	glm::vec3* dL_dmeans,
	float* dL_dcolor,
	float* dL_dcov3D,
	float* dL_dsh,
	glm::vec3* dL_dscale,
	glm::vec4* dL_drot)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P || !(radii[idx] > 0))
		return;

	float3 m = means[idx];

	// Taking care of gradients from the screenspace points
	// float4 m_hom = transformPoint4x4(m, proj);
	// float m_w = 1.0f / (m_hom.w + 0.0000001f);
	float2 mean_bev = compute3DtoBEV(m, proj);


	// Compute loss gradient w.r.t. 3D means due to gradients of 2D means
	// from rendering procedure
	glm::vec3 dL_dmean;

	dL_dmean.x = dL_dmean2D[idx].x * proj[0] + dL_dmean2D[idx].y * proj[4];
	dL_dmean.y = dL_dmean2D[idx].x * proj[1] + dL_dmean2D[idx].y * proj[5];
	dL_dmean.z = 0;
	// That's the second part of the mean gradient. Previous computation
	// of cov2D and following SH conversion also affects it.
	dL_dmeans[idx] += dL_dmean;

	// Compute gradient updates due to computing covariance from scale/rotation
	if (scales)
		computeCov3D(idx, scales[idx], scale_modifier, rotations[idx], dL_dcov3D, dL_dscale, dL_drot);
}

// Backward version of the rendering procedure.
template <uint32_t C>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	const float* __restrict__ bg_color,
	const float2* __restrict__ points_xy_image,
	const float4* __restrict__ conic_opacity,
	const float* __restrict__ colors,
	const float* __restrict__ final_Ts,
	const uint32_t* __restrict__ n_contrib,
	const float* __restrict__ dL_dpixels,
	float3* __restrict__ dL_dmean2D,
	float4* __restrict__ dL_dconic2D,
	float* __restrict__ dL_dopacity,
	float* __restrict__ dL_dcolors)
{
	// We rasterize again. Compute necessary block info.
	auto block = cg::this_thread_block();
	const uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	const uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	const uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	const uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	const uint32_t pix_id = W * pix.y + pix.x;
	const float2 pixf = { (float)pix.x, (float)pix.y };

	const bool inside = pix.x < W&& pix.y < H;
	const uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];

	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);

	bool done = !inside;
	int toDo = range.y - range.x;

	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];
	__shared__ float collected_colors[C * BLOCK_SIZE];

	// In the forward, we stored the final value for T, the
	// product of all (1 - alpha) factors. 
	const float T_final = inside ? final_Ts[pix_id] : 0;
	float T = T_final;

	// We start from the back. The ID of the last contributing
	// Gaussian is known from each pixel from the forward.
	uint32_t contributor = toDo;
	const int last_contributor = inside ? n_contrib[pix_id] : 0;

	float accum_rec[C] = { 0 };
	float dL_dpixel[C];
	if (inside)
		for (int i = 0; i < C; i++)
			dL_dpixel[i] = dL_dpixels[i * H * W + pix_id];

	float last_alpha = 0;
	float last_color[C] = { 0 };

	// Gradient of pixel coordinate w.r.t. normalized 
	// screen-space viewport corrdinates (-1 to 1)
	const float ddelx_dx = 0.5 * W;
	const float ddely_dy = 0.5 * H;

	// Traverse all Gaussians
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// Load auxiliary data into shared memory, start in the BACK
		// and load them in revers order.
		block.sync();
		const int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			const int coll_id = point_list[range.y - progress - 1];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
			for (int i = 0; i < C; i++)
				collected_colors[i * BLOCK_SIZE + block.thread_rank()] = colors[coll_id * C + i];
		}
		block.sync();

		// Iterate over Gaussians
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current Gaussian ID. Skip, if this one
			// is behind the last contributor for this pixel.
			contributor--;
			if (contributor >= last_contributor)
				continue;

			// Compute blending values, as before.
			const float2 xy = collected_xy[j];
			const float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			const float4 con_o = collected_conic_opacity[j];
			const float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f)
				continue;

			const float G = exp(power);
			const float alpha = min(0.99f, con_o.w * G);
			if (alpha < 1.0f / 255.0f)
				continue;

			T = T / (1.f - alpha);
			const float dchannel_dcolor = alpha * T;

			// Propagate gradients to per-Gaussian colors and keep
			// gradients w.r.t. alpha (blending factor for a Gaussian/pixel
			// pair).
			float dL_dalpha = 0.0f;
			const int global_id = collected_id[j];
			for (int ch = 0; ch < C; ch++)
			{
				const float c = collected_colors[ch * BLOCK_SIZE + j];
				// Update last color (to be used in the next iteration)
				accum_rec[ch] = last_alpha * last_color[ch] + (1.f - last_alpha) * accum_rec[ch];
				last_color[ch] = c;

				const float dL_dchannel = dL_dpixel[ch];
				dL_dalpha += (c - accum_rec[ch]) * dL_dchannel;
				// Update the gradients w.r.t. color of the Gaussian. 
				// Atomic, since this pixel is just one of potentially
				// many that were affected by this Gaussian.
				atomicAdd(&(dL_dcolors[global_id * C + ch]), dchannel_dcolor * dL_dchannel);
			}
			dL_dalpha *= T;
			// Update last alpha (to be used in the next iteration)
			last_alpha = alpha;

			// Account for fact that alpha also influences how much of
			// the background color is added if nothing left to blend
			float bg_dot_dpixel = 0;
			for (int i = 0; i < C; i++)
				bg_dot_dpixel += bg_color[i] * dL_dpixel[i];
			dL_dalpha += (-T_final / (1.f - alpha)) * bg_dot_dpixel;


			// Helpful reusable temporary variables
			const float dL_dG = con_o.w * dL_dalpha;
			const float gdx = G * d.x;
			const float gdy = G * d.y;
			const float dG_ddelx = -gdx * con_o.x - gdy * con_o.y;
			const float dG_ddely = -gdy * con_o.z - gdx * con_o.y;

			// Update gradients w.r.t. 2D mean position of the Gaussian
			atomicAdd(&dL_dmean2D[global_id].x, dL_dG * dG_ddelx * ddelx_dx);
			atomicAdd(&dL_dmean2D[global_id].y, dL_dG * dG_ddely * ddely_dy);

			// Update gradients w.r.t. 2D covariance (2x2 matrix, symmetric)
			atomicAdd(&dL_dconic2D[global_id].x, -0.5f * gdx * d.x * dL_dG);
			atomicAdd(&dL_dconic2D[global_id].y, -0.5f * gdx * d.y * dL_dG);
			atomicAdd(&dL_dconic2D[global_id].w, -0.5f * gdy * d.y * dL_dG);

			// Update gradients w.r.t. opacity of the Gaussian
			atomicAdd(&(dL_dopacity[global_id]), G * dL_dalpha);
		}
	}
}

void BACKWARD::preprocess(
	int P, int D, int M,
	const float3* means3D,
	const int* radii,
	const float* shs,
	const bool* clamped,
	const glm::vec3* scales,
	const glm::vec4* rotations,
	const float scale_modifier,
	const float* cov3Ds,
	const float* viewmatrix,
	const float* projmatrix,
	// const float focal_x, float focal_y,
	// const float tan_fovx, float tan_fovy,
	// const glm::vec3* campos,
	const float3* dL_dmean2D,
	const float* dL_dconic,
	glm::vec3* dL_dmean3D,
	float* dL_dcolor,
	float* dL_dcov3D,
	float* dL_dsh,
	glm::vec3* dL_dscale,
	glm::vec4* dL_drot)
{
	// Propagate gradients for the path of 2D conic matrix computation. 
	// Somewhat long, thus it is its own kernel rather than being part of 
	// "preprocess". When done, loss gradient w.r.t. 3D means has been
	// modified and gradient w.r.t. 3D covariance matrix has been computed.	
	computeCov2DBEVCUDA << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		radii,
		cov3Ds,
		// focal_x,
		// focal_y,
		// tan_fovx,
		// tan_fovy,
		viewmatrix,
		dL_dconic,
		// (float3*)dL_dmean3D,
		dL_dcov3D);

	// Propagate gradients for remaining steps: finish 3D mean gradients,
	// propagate color gradients to SH (if desireD), propagate 3D covariance
	// matrix gradients to scale and rotation.
	preprocessCUDA<NUM_CHANNELS> << < (P + 255) / 256, 256 >> > (
		P, D, M,
		(float3*)means3D,
		radii,
		shs,
		clamped,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		projmatrix,
		// campos,
		(float3*)dL_dmean2D,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_dcov3D,
		dL_dsh,
		dL_dscale,
		dL_drot);
}

void BACKWARD::render(
	const dim3 grid, const dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	const float* bg_color,
	const float2* means2D,
	const float4* conic_opacity,
	const float* colors,
	const float* final_Ts,
	const uint32_t* n_contrib,
	const float* dL_dpixels,
	float3* dL_dmean2D,
	float4* dL_dconic2D,
	float* dL_dopacity,
	float* dL_dcolors)
{
	renderCUDA<NUM_CHANNELS> << <grid, block >> >(
		ranges,
		point_list,
		W, H,
		bg_color,
		means2D,
		conic_opacity,
		colors,
		final_Ts,
		n_contrib,
		dL_dpixels,
		dL_dmean2D,
		dL_dconic2D,
		dL_dopacity,
		dL_dcolors
		);
}